#include "hip/hip_runtime.h"
#include <simulation_methods.h>


typedef struct RKF45StepResult {
    PendulumState pendulumState;
    FloatType timeStepUsedInCalculation;
    FloatType newTimeStep;
} RKF45StepResult;


__constant__ FloatType kScalesList[6][5] = {{0,0,0,0,0},
                                            {1.0/4.0,0,0,0,0},
                                            {3.0/32.0,9.0/32.0,0,0,0},
                                            {1932.0/2197.0,-7200.0/2197.0,7296.0/2197.0,0,0},
                                            {439.0/216.0,-8.0,3680.0/513.0,-845.0/4104.0,0},
                                            {-8.0/27.0,2.0,-3544.0/2565.0,1859.0/4104.0,-11.0/40.0}};
__constant__ FloatType fourthOrderConstants[4] = {25.0/216.0, 1408.0/2565.0, 2197.0/4104.0, -1.0/5.0};
__constant__ FloatType fifthOrderConstants[5] = {16.0/135.0, 6656.0/12825.0, 28561.0/56430.0, -9.0/50.0, 2.0/55.0};


__device__ void compute_rkf_step(PendulumState pendulumState,
                                 FloatType u,
                                 FloatType length1, FloatType length2,
                                 FloatType g,
                                 FloatType kList[6][4], int kListSize, FloatType* kScales,
                                 FloatType timeStep) {

    // Compute the new pendulum state using Forward Euler using every k element.
    FloatType kSums[4] = {0,0,0,0};
    for (int i = 0; i < kListSize; i++) {
        for (int j = 0; j < 4; j++) {
            kSums[j] += kList[i][j]*kScales[i];
        }
    }

    PendulumState newPendulumState;
    newPendulumState.angle1 = pendulumState.angle1 + timeStep*kSums[0];
    newPendulumState.angle2 = pendulumState.angle2 + timeStep*kSums[1];
    newPendulumState.angularVelocity1 = pendulumState.angularVelocity1 + timeStep*kSums[2];
    newPendulumState.angularVelocity2 = pendulumState.angularVelocity2 + timeStep*kSums[3];

    // Compute the accelerations at the new pendulum state.
    AccelerationResults accelerationResults = compute_accelerations(newPendulumState, u, length1, length2, g);

    // Return the computed derivatives of position and velocity.
    kList[kListSize][0] = newPendulumState.angularVelocity1;
    kList[kListSize][1] = newPendulumState.angularVelocity2;
    kList[kListSize][2] = accelerationResults.acceleration1;
    kList[kListSize][3] = accelerationResults.acceleration2;
}


__device__ RKF45StepResult compute_double_pendulum_step_rkf45(PendulumState pendulumState,
                                                              FloatType u,
                                                              FloatType length1, FloatType length2,
                                                              FloatType g,
                                                              FloatType timeStep, FloatType errorTolerance) {

    FloatType kList[6][4] = {{0,0,0,0},
                             {0,0,0,0},
                             {0,0,0,0},
                             {0,0,0,0},
                             {0,0,0,0},
                             {0,0,0,0}};

    // Keep recalculating the step with a smaller time step until the given error tolerance is reached.
    while(1) {
        // Compute K values.
        for (int i = 0; i < 6; i++) {
            compute_rkf_step(pendulumState, u, length1, length2, g, kList, i, kScalesList[i], timeStep);
        }

        // Compute the new state of the pendulum with 4th and 5th order methods, and compute what the new time step should be.
        PendulumState newPendulumState;
        FloatType* pendulumStateValues = &(pendulumState.angle1);
        FloatType* newPendulumStateValues = &(newPendulumState.angle1);
        bool stepNeedsToBeRecalculated = false;
        FloatType timeStepToUseInRecalculation = 2*timeStep;
        FloatType timeStepToUseInNextStep = 2*timeStep;
        for (int i = 0; i < 4; i++) {
            // Compute the value of the variable after one step with 4th and 5th order methods.
            FloatType cur4thOrderResult = pendulumStateValues[i] + (fourthOrderConstants[0]*kList[0][i] + fourthOrderConstants[1]*kList[2][i] + fourthOrderConstants[2]*kList[3][i] + fourthOrderConstants[3]*kList[4][i])*timeStep;
            FloatType cur5thOrderResult = pendulumStateValues[i] + (fifthOrderConstants[0]*kList[0][i] + fifthOrderConstants[1]*kList[2][i] + fifthOrderConstants[2]*kList[3][i] + fifthOrderConstants[3]*kList[4][i] + fifthOrderConstants[4]*kList[5][i])*timeStep;
            newPendulumStateValues[i] = cur4thOrderResult;

            // Compute what the new time step should be. The smallest new time step computed for the four pendulum state variables is used.
            if (cur4thOrderResult != cur5thOrderResult) {
                FloatType R = abs(cur4thOrderResult - cur5thOrderResult) / timeStep;
                FloatType delta = .84*pow(errorTolerance/R, (FloatType).25);
                FloatType curTimeStepToUseInNextStep = delta*timeStep;
                timeStepToUseInNextStep = min(timeStepToUseInNextStep, curTimeStepToUseInNextStep);

                // If R is greater than the error tolerance then recompute the step with a smaller step size.
                if (R > errorTolerance) {
                    stepNeedsToBeRecalculated = true;
                    timeStepToUseInRecalculation = min(timeStepToUseInRecalculation, curTimeStepToUseInNextStep);
                }
            }
        }

        // If the tolerance was met for all of the variables then return the result.
        if (!stepNeedsToBeRecalculated) {
            RKF45StepResult result;
            result.pendulumState = newPendulumState;
            result.timeStepUsedInCalculation = timeStep;
            result.newTimeStep = timeStepToUseInNextStep;
            return result;
        }

        timeStep = timeStepToUseInNextStep;
    }
}


__global__ void compute_double_pendulum_fractal_time_till_flip_from_initial_states(FloatType m1, FloatType m2,
                                                                                   FloatType length1, FloatType length2,
                                                                                   FloatType g,
                                                                                   FloatType angle1Min, FloatType angle1Max,
                                                                                   FloatType angle2Min, FloatType angle2Max,
                                                                                   PendulumState *pendulumStates,
                                                                                   bool startFromDefaultState,
                                                                                   FloatType amountOfTimeAlreadyExecuted,
                                                                                   int totalNumberOfAnglesToTestX, int totalNumberOfAnglesToTestY,
                                                                                   FloatType timeStep,
                                                                                   FloatType errorTolerance,
                                                                                   FloatType maxTimeToSeeIfPendulumFlips,
                                                                                   FloatType *timeTillFlip) {

    int stepX = gridDim.x*blockDim.x;
    int stepY =  gridDim.y*blockDim.y;

    int startX = threadIdx.x + blockDim.x*blockIdx.x;
    int startY = threadIdx.y + blockDim.y*blockIdx.y;

    // Pre-compute a commonly used value.
    FloatType u = 1 + m1/m2;

    // Simulate the double pendulums.
    for (int x = startX; x < totalNumberOfAnglesToTestX; x += stepX) {
        for (int y = startY; y < totalNumberOfAnglesToTestY; y += stepY) {
            int pixelIndex = (totalNumberOfAnglesToTestY - y - 1)*totalNumberOfAnglesToTestX + x;

            // Set the initial state of the pendulum for the current pixel.
            PendulumState initialPendulumState;
            if (startFromDefaultState) {
                initialPendulumState.angle1 = angle1Min + FloatType(x)*(angle1Max - angle1Min)/FloatType(totalNumberOfAnglesToTestX - 1);
                initialPendulumState.angle2 = angle2Min + FloatType(y)*(angle2Max - angle2Min)/FloatType(totalNumberOfAnglesToTestY - 1);
                initialPendulumState.angularVelocity1 = 0;
                initialPendulumState.angularVelocity2 = 0;
            }
            else {
                initialPendulumState.angle1 = pendulumStates[pixelIndex].angle1;
                initialPendulumState.angle2 = pendulumStates[pixelIndex].angle2;
                initialPendulumState.angularVelocity1 = pendulumStates[pixelIndex].angularVelocity1;
                initialPendulumState.angularVelocity2 = pendulumStates[pixelIndex].angularVelocity2;
            }

            // If not given initial states, skip the current pendulum if it doesn't have enough initial energy to
            // flip the first mass.
            if (startFromDefaultState) {
                Point point1Position = get_point_position({0,0}, initialPendulumState.angle1, length1);
                Point point2Position = get_point_position(point1Position, initialPendulumState.angle2, length2);
                FloatType potentialEnergy1 = point1Position.y*m1*g;
                FloatType potentialEnergy2 = point2Position.y*m2*g;
                FloatType totalPotentialEnergy = potentialEnergy1 + potentialEnergy2;
                FloatType minimumEnergyNeededForFlip = m1*length1*g + m2*(length1 - length2)*g;
                if (totalPotentialEnergy < minimumEnergyNeededForFlip) {
                    timeTillFlip[pixelIndex] = NotEnoughEnergyToFlip;
                    continue;
                }
            }

            // Otherwise skip the pendulum if the number of current time steps at the current pendulum is -1, indicating
            // it originally didn't have enough energy to flip, or the pendulum already flipped.
            else if (timeTillFlip[pixelIndex] == NotEnoughEnergyToFlip ||
                     timeTillFlip[pixelIndex] != DidNotFlip) {
                continue;
            }

            // Simulate the pendulum until it flips or time runs out.
            PendulumState pendulumState = initialPendulumState;
            FloatType originalAngle1 = pendulumState.angle1;
            FloatType totalTimeExecuted = amountOfTimeAlreadyExecuted;
            bool pendulumFlipped = false;
            while (totalTimeExecuted < maxTimeToSeeIfPendulumFlips) {
                // Compute one time step of the pendulum simulation.
                RKF45StepResult result = compute_double_pendulum_step_rkf45(pendulumState, u, length1, length2, g, timeStep, errorTolerance);
                pendulumState = result.pendulumState;
                totalTimeExecuted += result.timeStepUsedInCalculation;
                timeStep = result.newTimeStep;

                // Check to see if the first mass flipped.
                if (floor((pendulumState.angle1 - PI) / TAU) != floor((originalAngle1 - PI) / TAU)) {
                    pendulumFlipped = true;
                    break;
                }
                originalAngle1 = pendulumState.angle1;
            }

            // Set the new number of time steps for the pendulum to flip, and the new pendulum state.
            // Set the number of time steps to -2 if it didn't flip.
            timeTillFlip[pixelIndex] = pendulumFlipped ? totalTimeExecuted : DidNotFlip;
            pendulumStates[pixelIndex] = pendulumState;
        }
    }
}


__global__ void compute_colors_from_time_till_flip(FloatType *timeTillFlip,
                                                   char *colors,
                                                   int totalNumberOfAnglesToTestX,
                                                   int totalNumberOfAnglesToTestY,
                                                   FloatType redScale,
                                                   FloatType greenScale,
                                                   FloatType blueScale,
                                                   FloatType shift) {

    int stepX = gridDim.x*blockDim.x;
    int stepY =  gridDim.y*blockDim.y;

    int startX = threadIdx.x + blockDim.x*blockIdx.x;
    int startY = threadIdx.y + blockDim.y*blockIdx.y;

    int area = totalNumberOfAnglesToTestX*totalNumberOfAnglesToTestY;
    FloatType colorScales[] = {redScale, greenScale, blueScale};

    // Compute the color of each pixel.
    for (int x = startX; x < totalNumberOfAnglesToTestX; x += stepX) {
        for (int y = startY; y < totalNumberOfAnglesToTestY; y += stepY) {
            int pixelIndex = (totalNumberOfAnglesToTestY - y - 1)*totalNumberOfAnglesToTestX + x;
            FloatType curTimeTillFlip = timeTillFlip[pixelIndex];

            // Compute the color of the sample. Color it black if the pendulum did not flip.
            FloatType timeTillFlipMs = curTimeTillFlip*1000.0;
            if (curTimeTillFlip == NotEnoughEnergyToFlip || curTimeTillFlip == DidNotFlip) {
                timeTillFlipMs = 0;
            }
            for (int i = 0; i < 3; i++) {
                colors[pixelIndex + i*area] = lroundf(abs(sin(1.0/255 * PI * timeTillFlipMs * colorScales[i] * shift)) * 255);
            }
        }
    }
}



