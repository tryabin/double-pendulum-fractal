#include "hip/hip_runtime.h"
#include <simulation_methods.h>

// Runge-Kutta-Fehlberg Butcher tableau constants
#ifdef RKF45
    __constant__ FloatType butcherTableau[6][5] = {{0,0,0,0,0},
                                                      {1.0/4.0,0,0,0,0},
                                                      {3.0/32.0,9.0/32.0,0,0,0},
                                                      {1932.0/2197.0,-7200.0/2197.0,7296.0/2197.0,0,0},
                                                      {439.0/216.0,-8.0,3680.0/513.0,-845.0/4104.0,0},
                                                      {-8.0/27.0,2.0,-3544.0/2565.0,1859.0/4104.0,-11.0/40.0}};
    __constant__ FloatType rkFourthOrderConstants[4] = {25.0/216.0, 1408.0/2565.0, 2197.0/4104.0, -1.0/5.0};
    __constant__ FloatType rkFifthOrderConstants[5] = {16.0/135.0, 6656.0/12825.0, 28561.0/56430.0, -9.0/50.0, 2.0/55.0};

// Cash-Karp Butcher tableau constants
#elif CASH_KARP
    __constant__ FloatType butcherTableau[6][5] = {{0,0,0,0,0},
                                                   {1.0/5.0,0,0,0,0},
                                                   {3.0/40.0,9.0/40.0,0,0,0},
                                                   {3.0/10.0,-9.0/10.0,6.0/5.0,0,0},
                                                   {-11.0/54.0,5.0/2.0,-70.0/27.0,35.0/27.0,0},
                                                   {1631.0/55296.0,175.0/512,575.0/13824.0,44275.0/110592.0,253.0/4096.0}};
    __constant__ FloatType rkFourthOrderConstants[5] = {2825.0/27648.0, 18575.0/48384.0, 13525.0/55296.0, 277.0/14336.0, 1.0/4.0};
    __constant__ FloatType rkFifthOrderConstants[4] = {37.0/378.0, 250.0/621.0, 125.0/594.0, 512.0/1771.0};
#else
    #error Adaptive step-size method not provided
#endif


typedef struct AdaptiveStepSizeResult {
    PendulumState pendulumState;
    FloatType timeStepUsedInCalculation;
    FloatType newTimeStep;
} AdaptiveStepSizeResult;


__device__ void compute_step(PendulumState pendulumState,
                             FloatType u,
                             FloatType length1, FloatType length2,
                             FloatType g,
                             FloatType kList[6][4], int kListSize, FloatType* kScales,
                             FloatType timeStep) {

    // Compute the new pendulum state using Forward Euler using every k element.
    FloatType kSums[4] = {0,0,0,0};
    for (int i = 0; i < kListSize; i++) {
        for (int j = 0; j < 4; j++) {
            kSums[j] += kList[i][j]*kScales[i];
        }
    }

    PendulumState newPendulumState;
    newPendulumState.angle1 = pendulumState.angle1 + timeStep*kSums[0];
    newPendulumState.angle2 = pendulumState.angle2 + timeStep*kSums[1];
    newPendulumState.angularVelocity1 = pendulumState.angularVelocity1 + timeStep*kSums[2];
    newPendulumState.angularVelocity2 = pendulumState.angularVelocity2 + timeStep*kSums[3];

    // Compute the accelerations at the new pendulum state.
    AccelerationResults accelerationResults = compute_accelerations(newPendulumState, u, length1, length2, g);

    // Return the computed derivatives of position and velocity.
    kList[kListSize][0] = newPendulumState.angularVelocity1;
    kList[kListSize][1] = newPendulumState.angularVelocity2;
    kList[kListSize][2] = accelerationResults.acceleration1;
    kList[kListSize][3] = accelerationResults.acceleration2;
}


__device__ AdaptiveStepSizeResult compute_double_pendulum_adaptive_step_size_method(PendulumState pendulumState,
                                                                                    FloatType u,
                                                                                    FloatType length1, FloatType length2,
                                                                                    FloatType g,
                                                                                    FloatType timeStep, FloatType errorTolerance) {

    FloatType kList[6][4] = {{0,0,0,0},
                             {0,0,0,0},
                             {0,0,0,0},
                             {0,0,0,0},
                             {0,0,0,0},
                             {0,0,0,0}};

    // Keep recalculating the step with a smaller time step until the given error tolerance is reached.
    while(1) {
        // Compute K values.
        for (int i = 0; i < 6; i++) {
            compute_step(pendulumState, u, length1, length2, g, kList, i, butcherTableau[i], timeStep);
        }

        // Compute the new state of the pendulum with 4th and 5th order methods, and compute what the new time step should be.
        PendulumState newPendulumState;
        FloatType* pendulumStateValues = &(pendulumState.angle1);
        FloatType* newPendulumStateValues = &(newPendulumState.angle1);
        bool stepNeedsToBeRecalculated = false;
        FloatType timeStepToUseInRecalculation = 2*timeStep;
        FloatType timeStepToUseInNextStep = 2*timeStep;
        for (int i = 0; i < 4; i++) {
            // Compute the value of the variable after one step with 4th and 5th order methods.
            // Different methods are compiled depending on the chosen algorithm.
            #ifdef RKF45
                FloatType cur4thOrderResult = pendulumStateValues[i] + (rkFourthOrderConstants[0]*kList[0][i] + rkFourthOrderConstants[1]*kList[2][i] + rkFourthOrderConstants[2]*kList[3][i] + rkFourthOrderConstants[3]*kList[4][i])*timeStep;
                FloatType cur5thOrderResult = pendulumStateValues[i] + (rkFifthOrderConstants[0]*kList[0][i] + rkFifthOrderConstants[1]*kList[2][i] + rkFifthOrderConstants[2]*kList[3][i] + rkFifthOrderConstants[3]*kList[4][i] + rkFifthOrderConstants[4]*kList[5][i])*timeStep;
                newPendulumStateValues[i] = cur4thOrderResult;
            #elif CASH_KARP
                FloatType cur4thOrderResult = pendulumStateValues[i] + (rkFourthOrderConstants[0]*kList[0][i] + rkFourthOrderConstants[1]*kList[2][i] + rkFourthOrderConstants[2]*kList[3][i] + rkFourthOrderConstants[3]*kList[4][i] + rkFourthOrderConstants[4]*kList[5][i])*timeStep;
                FloatType cur5thOrderResult = pendulumStateValues[i] + (rkFifthOrderConstants[0]*kList[0][i] + rkFifthOrderConstants[1]*kList[2][i] + rkFifthOrderConstants[2]*kList[3][i] + rkFifthOrderConstants[3]*kList[5][i])*timeStep;
                newPendulumStateValues[i] = cur4thOrderResult;
            #endif

            // Compute what the new time step should be. The smallest new time step computed for the four pendulum state variables is used.
            if (cur4thOrderResult != cur5thOrderResult) {
                FloatType R = abs(cur4thOrderResult - cur5thOrderResult) / timeStep;
                FloatType delta = .84*sqrt(sqrt(errorTolerance/R));
                FloatType curTimeStepToUseInNextStep = delta*timeStep;
                timeStepToUseInNextStep = min(timeStepToUseInNextStep, curTimeStepToUseInNextStep);

                // If R is greater than the error tolerance then recompute the step with a smaller step size.
                if (R > errorTolerance) {
                    stepNeedsToBeRecalculated = true;
                    timeStepToUseInRecalculation = min(timeStepToUseInRecalculation, curTimeStepToUseInNextStep);
                }
            }
        }

        // If the tolerance was met for all of the variables then return the result.
        if (!stepNeedsToBeRecalculated) {
            AdaptiveStepSizeResult result;
            result.pendulumState = newPendulumState;
            result.timeStepUsedInCalculation = timeStep;
            result.newTimeStep = timeStepToUseInNextStep;
            return result;
        }

        timeStep = timeStepToUseInNextStep;
    }
}


__global__ void compute_double_pendulum_fractal_time_till_flip_from_initial_states(FloatType m1, FloatType m2,
                                                                                   FloatType length1, FloatType length2,
                                                                                   FloatType g,
                                                                                   FloatType angle1Min, FloatType angle1Max,
                                                                                   FloatType angle2Min, FloatType angle2Max,
                                                                                   PendulumState *pendulumStates,
                                                                                   bool startFromDefaultState,
                                                                                   FloatType amountOfTimeAlreadyExecuted,
                                                                                   int totalNumberOfAnglesToTestX, int totalNumberOfAnglesToTestY,
                                                                                   FloatType timeStep,
                                                                                   FloatType errorTolerance,
                                                                                   FloatType maxTimeToSeeIfPendulumFlips,
                                                                                   FloatType *timeTillFlip) {

    int stepX = gridDim.x*blockDim.x;
    int stepY =  gridDim.y*blockDim.y;

    int startX = threadIdx.x + blockDim.x*blockIdx.x;
    int startY = threadIdx.y + blockDim.y*blockIdx.y;

    // Pre-compute a commonly used value.
    FloatType u = 1 + m1/m2;

    // Simulate the double pendulums.
    for (int x = startX; x < totalNumberOfAnglesToTestX; x += stepX) {
        for (int y = startY; y < totalNumberOfAnglesToTestY; y += stepY) {
            int pixelIndex = (totalNumberOfAnglesToTestY - y - 1)*totalNumberOfAnglesToTestX + x;

            // Set the initial state of the pendulum for the current pixel.
            PendulumState initialPendulumState;
            if (startFromDefaultState) {
                initialPendulumState.angle1 = angle1Min + FloatType(x)*(angle1Max - angle1Min)/FloatType(totalNumberOfAnglesToTestX - 1);
                initialPendulumState.angle2 = angle2Min + FloatType(y)*(angle2Max - angle2Min)/FloatType(totalNumberOfAnglesToTestY - 1);
                initialPendulumState.angularVelocity1 = 0;
                initialPendulumState.angularVelocity2 = 0;
            }
            else {
                initialPendulumState.angle1 = pendulumStates[pixelIndex].angle1;
                initialPendulumState.angle2 = pendulumStates[pixelIndex].angle2;
                initialPendulumState.angularVelocity1 = pendulumStates[pixelIndex].angularVelocity1;
                initialPendulumState.angularVelocity2 = pendulumStates[pixelIndex].angularVelocity2;
            }

            // If not given initial states, skip the current pendulum if it doesn't have enough initial energy to
            // flip the first mass.
            if (startFromDefaultState) {
                Point point1Position = get_point_position({0,0}, initialPendulumState.angle1, length1);
                Point point2Position = get_point_position(point1Position, initialPendulumState.angle2, length2);
                FloatType potentialEnergy1 = point1Position.y*m1*g;
                FloatType potentialEnergy2 = point2Position.y*m2*g;
                FloatType totalPotentialEnergy = potentialEnergy1 + potentialEnergy2;
                FloatType minimumEnergyNeededForFlip = m1*length1*g + m2*(length1 - length2)*g;
                if (totalPotentialEnergy < minimumEnergyNeededForFlip) {
                    timeTillFlip[pixelIndex] = NotEnoughEnergyToFlip;
                    continue;
                }
            }

            // Otherwise skip the pendulum if the number of current time steps at the current pendulum is -1, indicating
            // it originally didn't have enough energy to flip, or the pendulum already flipped.
            else if (timeTillFlip[pixelIndex] == NotEnoughEnergyToFlip ||
                     timeTillFlip[pixelIndex] != DidNotFlip) {
                continue;
            }

            // Simulate the pendulum until it flips or time runs out.
            PendulumState pendulumState = initialPendulumState;
            FloatType originalAngle1 = pendulumState.angle1;
            FloatType totalTimeExecuted = amountOfTimeAlreadyExecuted;
            bool pendulumFlipped = false;
            while (totalTimeExecuted < maxTimeToSeeIfPendulumFlips) {
                // Compute one time step of the pendulum simulation.
                AdaptiveStepSizeResult result = compute_double_pendulum_adaptive_step_size_method(pendulumState, u, length1, length2, g, timeStep, errorTolerance);
                pendulumState = result.pendulumState;
                totalTimeExecuted += result.timeStepUsedInCalculation;
                timeStep = result.newTimeStep;

                // Check to see if the first mass flipped.
                if (floor((pendulumState.angle1 - PI) / TAU) != floor((originalAngle1 - PI) / TAU)) {
                    pendulumFlipped = true;
                    break;
                }
                originalAngle1 = pendulumState.angle1;
            }

            // Set the new number of time steps for the pendulum to flip, and the new pendulum state.
            // Set the number of time steps to -2 if it didn't flip.
            timeTillFlip[pixelIndex] = pendulumFlipped ? totalTimeExecuted : DidNotFlip;
            pendulumStates[pixelIndex] = pendulumState;
        }
    }
}


__global__ void compute_colors_from_time_till_flip(FloatType *timeTillFlip,
                                                   char *colors,
                                                   int totalNumberOfAnglesToTestX,
                                                   int totalNumberOfAnglesToTestY,
                                                   FloatType redScale,
                                                   FloatType greenScale,
                                                   FloatType blueScale,
                                                   FloatType shift) {

    int stepX = gridDim.x*blockDim.x;
    int stepY =  gridDim.y*blockDim.y;

    int startX = threadIdx.x + blockDim.x*blockIdx.x;
    int startY = threadIdx.y + blockDim.y*blockIdx.y;

    int area = totalNumberOfAnglesToTestX*totalNumberOfAnglesToTestY;
    FloatType colorScales[] = {redScale, greenScale, blueScale};

    // Compute the color of each pixel.
    for (int x = startX; x < totalNumberOfAnglesToTestX; x += stepX) {
        for (int y = startY; y < totalNumberOfAnglesToTestY; y += stepY) {
            int pixelIndex = (totalNumberOfAnglesToTestY - y - 1)*totalNumberOfAnglesToTestX + x;
            FloatType curTimeTillFlip = timeTillFlip[pixelIndex];

            // Compute the color of the sample. Color it black if the pendulum did not flip.
            FloatType timeTillFlipMs = curTimeTillFlip*1000.0;
            if (curTimeTillFlip == NotEnoughEnergyToFlip || curTimeTillFlip == DidNotFlip) {
                timeTillFlipMs = 0;
            }
            for (int i = 0; i < 3; i++) {
                colors[pixelIndex + i*area] = lroundf(abs(sin(1.0/255 * PI * timeTillFlipMs * colorScales[i] * shift)) * 255);
            }
        }
    }
}



