#include "hip/hip_runtime.h"
#include <simulation_methods.h>
#include <butcher_tableaus.h>


typedef struct AdaptiveStepSizeResult {
    PendulumState pendulumState;
    FloatType timeStepUsedInCalculation;
    FloatType newTimeStep;
} AdaptiveStepSizeResult;


__device__ void compute_step(PendulumState pendulumState,
                             FloatType u,
                             FloatType length1, FloatType length2,
                             FloatType g,
                             FloatType kList[12][4],
                             int butcherTableauRow,
                             FloatType timeStep) {

    // Compute the new pendulum state using Forward Euler using every k element.
    FloatType kSums[4] = {0,0,0,0};
    int startingButcherTableauIndex = butcherTableauRow*(butcherTableauRow - 1)/2;
    for (int i = 0; i < butcherTableauRow; i++) {
        for (int j = 0; j < 4; j++) {
            kSums[j] += kList[i][j]*butcherTableau[startingButcherTableauIndex + i];
        }
    }

    PendulumState newPendulumState;
    newPendulumState.angle1 = pendulumState.angle1 + timeStep*kSums[0];
    newPendulumState.angle2 = pendulumState.angle2 + timeStep*kSums[1];
    newPendulumState.angularVelocity1 = pendulumState.angularVelocity1 + timeStep*kSums[2];
    newPendulumState.angularVelocity2 = pendulumState.angularVelocity2 + timeStep*kSums[3];

    // Compute the accelerations at the new pendulum state.
    AccelerationResults accelerationResults = compute_accelerations(newPendulumState, u, length1, length2, g);

    // Return the computed derivatives of position and velocity.
    kList[butcherTableauRow][0] = newPendulumState.angularVelocity1;
    kList[butcherTableauRow][1] = newPendulumState.angularVelocity2;
    kList[butcherTableauRow][2] = accelerationResults.acceleration1;
    kList[butcherTableauRow][3] = accelerationResults.acceleration2;
}


__device__ AdaptiveStepSizeResult compute_double_pendulum_step_with_adaptive_step_size_method(PendulumState pendulumState,
                                                                                              FloatType u,
                                                                                              FloatType length1, FloatType length2,
                                                                                              FloatType g,
                                                                                              FloatType timeStep, FloatType errorTolerance) {

    // Keep recalculating the step with a smaller time step until the given error tolerance is reached.
    FloatType kList[12][4];
    while(1) {

        // Compute K values.
        #ifdef FEHLBERG_87
        for (int i = 0; i <= 12; i++) {
        #elif DORMAND_PRINCE_54
        for (int i = 0; i <= 6; i++) {
        #else
        for (int i = 0; i <= 5; i++) {
        #endif
            compute_step(pendulumState, u, length1, length2, g, kList, i, timeStep);
        }

        // Compute the new state of the pendulum with 4th and 5th order methods, and compute what the new time step should be.
        PendulumState newPendulumState;
        FloatType* pendulumStateValues = &(pendulumState.angle1);
        FloatType* newPendulumStateValues = &(newPendulumState.angle1);
        bool stepNeedsToBeRecalculated = false;
        FloatType timeStepToUseInRecalculation = 2*timeStep;
        FloatType timeStepToUseInNextStep = 2*timeStep;
        for (int i = 0; i < 4; i++) {
            // Compute the value of the variable after one step using the
            // adaptive step size method specified at compile time.
            #ifdef RKF_45
                FloatType curLowerOrderResult = pendulumStateValues[i] + (rkLowerOrderConstants[0]*kList[0][i] + rkLowerOrderConstants[1]*kList[2][i] + rkLowerOrderConstants[2]*kList[3][i] + rkLowerOrderConstants[3]*kList[4][i])*timeStep;
                FloatType curHigherOrderResult = pendulumStateValues[i] + (rkHigherOrderConstants[0]*kList[0][i] + rkHigherOrderConstants[1]*kList[2][i] + rkHigherOrderConstants[2]*kList[3][i] + rkHigherOrderConstants[3]*kList[4][i] + rkHigherOrderConstants[4]*kList[5][i])*timeStep;
                newPendulumStateValues[i] = curLowerOrderResult;
            #elif CASH_KARP_45
                FloatType curLowerOrderResult = pendulumStateValues[i] + (rkLowerOrderConstants[0]*kList[0][i] + rkLowerOrderConstants[1]*kList[2][i] + rkLowerOrderConstants[2]*kList[3][i] + rkLowerOrderConstants[3]*kList[4][i] + rkLowerOrderConstants[4]*kList[5][i])*timeStep;
                FloatType curHigherOrderResult = pendulumStateValues[i] + (rkHigherOrderConstants[0]*kList[0][i] + rkHigherOrderConstants[1]*kList[2][i] + rkHigherOrderConstants[2]*kList[3][i] + rkHigherOrderConstants[3]*kList[5][i])*timeStep;
                newPendulumStateValues[i] = curLowerOrderResult;
            #elif DORMAND_PRINCE_54
                FloatType curLowerOrderResult = pendulumStateValues[i] + (rkLowerOrderConstants[0]*kList[0][i] + rkLowerOrderConstants[1]*kList[2][i] + rkLowerOrderConstants[2]*kList[3][i] + rkLowerOrderConstants[3]*kList[4][i] + rkLowerOrderConstants[4]*kList[5][i] + rkLowerOrderConstants[5]*kList[6][i])*timeStep;
                FloatType curHigherOrderResult = pendulumStateValues[i] + (rkHigherOrderConstants[0]*kList[0][i] + rkHigherOrderConstants[1]*kList[2][i] + rkHigherOrderConstants[2]*kList[3][i] + rkHigherOrderConstants[3]*kList[4][i] + rkHigherOrderConstants[4]*kList[5][i])*timeStep;
                newPendulumStateValues[i] = curHigherOrderResult;
            #elif FEHLBERG_87
                FloatType curLowerOrderResult = pendulumStateValues[i] + (rkLowerOrderConstants[0]*kList[0][i] + rkLowerOrderConstants[1]*kList[5][i] + rkLowerOrderConstants[2]*kList[6][i] + rkLowerOrderConstants[3]*kList[7][i] + rkLowerOrderConstants[4]*kList[8][i] + rkLowerOrderConstants[5]*kList[9][i] + rkLowerOrderConstants[6]*kList[10][i])*timeStep;
                FloatType curHigherOrderResult = pendulumStateValues[i] + (rkHigherOrderConstants[0]*kList[5][i] + rkHigherOrderConstants[1]*kList[6][i] + rkHigherOrderConstants[2]*kList[7][i] + rkHigherOrderConstants[3]*kList[8][i] + rkHigherOrderConstants[4]*kList[9][i] + rkHigherOrderConstants[5]*kList[11][i] + rkHigherOrderConstants[6]*kList[12][i])*timeStep;
                newPendulumStateValues[i] = curHigherOrderResult;
            #endif

            // Compute what the new time step should be. The smallest new time step computed for the four pendulum state variables is used.
            if (curLowerOrderResult != curHigherOrderResult) {
                FloatType R = abs(curLowerOrderResult - curHigherOrderResult) / timeStep;
                #ifdef DORMAND_PRINCE_54
                    #ifdef FLOAT_32
                    FloatType delta = powf(errorTolerance/(2*R), 1.0/5.0);
                    #else
                    FloatType delta = pow(errorTolerance/(2*R), 1.0/5.0);
                    #endif
                #elif FEHLBERG_87
                    #ifdef FLOAT_32
                    FloatType delta = powf(errorTolerance/(2*R), 1.0/8.0);
                    #else
                    FloatType delta = pow(errorTolerance/(2*R), 1.0/8.0);
                    #endif
                #else
                    #ifdef FLOAT_32
                    FloatType delta = sqrtf(sqrtf(errorTolerance/(2*R)));
                    #else
                    FloatType delta = sqrt(sqrt(errorTolerance/(2*R)));
                    #endif
                #endif

                FloatType curTimeStepToUseInNextStep = delta*timeStep;
                timeStepToUseInNextStep = min(timeStepToUseInNextStep, curTimeStepToUseInNextStep);

                // If R is greater than the error tolerance then recompute the step with a smaller step size.
                if (R > errorTolerance) {
                    stepNeedsToBeRecalculated = true;
                    timeStepToUseInRecalculation = min(timeStepToUseInRecalculation, curTimeStepToUseInNextStep);
                }
            }
        }

        // If the tolerance was met for all of the variables then return the result.
        if (!stepNeedsToBeRecalculated) {
            AdaptiveStepSizeResult result;
            result.pendulumState = newPendulumState;
            result.timeStepUsedInCalculation = timeStep;
            result.newTimeStep = timeStepToUseInNextStep;
            return result;
        }

        timeStep = timeStepToUseInNextStep;
    }
}


__global__ void compute_double_pendulum_fractal_time_till_flip_from_initial_states(FloatType m1, FloatType m2,
                                                                                   FloatType length1, FloatType length2,
                                                                                   FloatType g,
                                                                                   FloatType angle1Min, FloatType angle1Max,
                                                                                   FloatType angle2Min, FloatType angle2Max,
                                                                                   PendulumState *pendulumStates,
                                                                                   bool startFromDefaultState,
                                                                                   FloatType amountOfTimeAlreadyExecuted,
                                                                                   int totalNumberOfAnglesToTestX, int totalNumberOfAnglesToTestY,
                                                                                   FloatType timeStep,
                                                                                   FloatType errorTolerance,
                                                                                   FloatType maxTimeToSeeIfPendulumFlips,
                                                                                   FloatType *timeTillFlip) {

    int stepX = gridDim.x*blockDim.x;
    int stepY =  gridDim.y*blockDim.y;

    int startX = threadIdx.x + blockDim.x*blockIdx.x;
    int startY = threadIdx.y + blockDim.y*blockIdx.y;

    // Pre-compute a commonly used value.
    FloatType u = 1 + m1/m2;

    // Simulate the double pendulums.
    for (int x = startX; x < totalNumberOfAnglesToTestX; x += stepX) {
        for (int y = startY; y < totalNumberOfAnglesToTestY; y += stepY) {
            int pixelIndex = (totalNumberOfAnglesToTestY - y - 1)*totalNumberOfAnglesToTestX + x;

            // Set the initial state of the pendulum for the current pixel.
            PendulumState initialPendulumState;
            if (startFromDefaultState) {
                initialPendulumState.angle1 = angle1Min + FloatType(x)*(angle1Max - angle1Min)/FloatType(totalNumberOfAnglesToTestX - 1);
                initialPendulumState.angle2 = angle2Min + FloatType(y)*(angle2Max - angle2Min)/FloatType(totalNumberOfAnglesToTestY - 1);
                initialPendulumState.angularVelocity1 = 0;
                initialPendulumState.angularVelocity2 = 0;
            }
            else {
                initialPendulumState.angle1 = pendulumStates[pixelIndex].angle1;
                initialPendulumState.angle2 = pendulumStates[pixelIndex].angle2;
                initialPendulumState.angularVelocity1 = pendulumStates[pixelIndex].angularVelocity1;
                initialPendulumState.angularVelocity2 = pendulumStates[pixelIndex].angularVelocity2;
            }

            // If starting from the default state, skip the current pendulum if it doesn't have enough initial energy to
            // flip the first mass.
            if (startFromDefaultState) {
                Point point1Position = get_point_position({0,0}, initialPendulumState.angle1, length1);
                Point point2Position = get_point_position(point1Position, initialPendulumState.angle2, length2);
                FloatType potentialEnergy1 = point1Position.y*m1*g;
                FloatType potentialEnergy2 = point2Position.y*m2*g;
                FloatType totalPotentialEnergy = potentialEnergy1 + potentialEnergy2;
                FloatType minimumEnergyNeededForFlip = m1*length1*g + m2*(length1 - length2)*g;
                if (totalPotentialEnergy < minimumEnergyNeededForFlip) {
                    timeTillFlip[pixelIndex] = NotEnoughEnergyToFlip;
                    continue;
                }
            }

            // Otherwise skip the pendulum if the time at the current pendulum is -1, indicating
            // it originally didn't have enough energy to flip, or -2, indicating that the pendulum already flipped.
            else if (timeTillFlip[pixelIndex] == NotEnoughEnergyToFlip ||
                     timeTillFlip[pixelIndex] != DidNotFlip) {
                continue;
            }

            // Simulate the pendulum until it flips or time runs out.
            PendulumState pendulumState = initialPendulumState;
            FloatType originalAngle1 = pendulumState.angle1;
            FloatType totalTimeExecuted = amountOfTimeAlreadyExecuted;
            bool pendulumFlipped = false;
            while (totalTimeExecuted < maxTimeToSeeIfPendulumFlips) {
                // Compute one time step of the pendulum simulation.
                AdaptiveStepSizeResult result = compute_double_pendulum_step_with_adaptive_step_size_method(pendulumState, u, length1, length2, g, timeStep, errorTolerance);
                pendulumState = result.pendulumState;
                totalTimeExecuted += result.timeStepUsedInCalculation;
                timeStep = result.newTimeStep;

                // Check to see if the first mass flipped.
                if (floor((pendulumState.angle1 - PI) / TAU) != floor((originalAngle1 - PI) / TAU)) {
                    pendulumFlipped = true;
                    break;
                }
                originalAngle1 = pendulumState.angle1;
            }

            // Set the new time for the pendulum to flip, and the new pendulum state.
            // Set the time to -2 if it didn't flip.
            timeTillFlip[pixelIndex] = pendulumFlipped ? totalTimeExecuted : DidNotFlip;
            pendulumStates[pixelIndex] = pendulumState;
        }
    }
}


__global__ void compute_colors_from_time_till_flip(FloatType *timeTillFlip,
                                                   char *colors,
                                                   int totalNumberOfAnglesToTestX,
                                                   int totalNumberOfAnglesToTestY,
                                                   FloatType redScale,
                                                   FloatType greenScale,
                                                   FloatType blueScale,
                                                   FloatType shift) {

    int stepX = gridDim.x*blockDim.x;
    int stepY =  gridDim.y*blockDim.y;

    int startX = threadIdx.x + blockDim.x*blockIdx.x;
    int startY = threadIdx.y + blockDim.y*blockIdx.y;

    int area = totalNumberOfAnglesToTestX*totalNumberOfAnglesToTestY;
    FloatType colorScales[] = {redScale, greenScale, blueScale};

    // Compute the color of each pixel.
    for (int x = startX; x < totalNumberOfAnglesToTestX; x += stepX) {
        for (int y = startY; y < totalNumberOfAnglesToTestY; y += stepY) {
            int pixelIndex = (totalNumberOfAnglesToTestY - y - 1)*totalNumberOfAnglesToTestX + x;
            FloatType curTimeTillFlip = timeTillFlip[pixelIndex];

            // Compute the color of the sample. Color it black if the pendulum did not flip.
            FloatType timeTillFlipMs = curTimeTillFlip*1000.0;
            if (curTimeTillFlip == NotEnoughEnergyToFlip || curTimeTillFlip == DidNotFlip) {
                timeTillFlipMs = 0;
            }
            for (int i = 0; i < 3; i++) {
                colors[pixelIndex + i*area] = lroundf(abs(sin(1.0/255 * PI * timeTillFlipMs * colorScales[i] * shift)) * 255);
            }
        }
    }
}



